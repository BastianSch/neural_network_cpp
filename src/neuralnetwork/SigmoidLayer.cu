#include "hip/hip_runtime.h"
#include <neuralnetwork/SigmoidLayer.h>

SigmoidLayer::SigmoidLayer(std::string name)
{
  this->name = name;
}

Matrix SigmoidLayer::forward(const Matrix& X)
{
  Matrix res = X;
    for(int i = 0; i < res.getRows(); i++)
    {
      for(int j = 0; j < res.getCols(); j++)
      {
        res(i, j) = 1/(1+exp(-res(i,j)));
      }
    }
    return X;
}

Matrix SigmoidLayer::backprop(Matrix& dZ, const Matrix& X, float learning_rate)
{
  // o(x)*(1-o(x))
  Matrix m1(X.getRows(), X.getCols(), 1.0f);
  Matrix ox = this->forward(dZ);
  m1 = m1 - ox;
  Matrix ox_T = ox.transpose();
  m1 = ox_T * m1;

  return m1.transpose();
}
#include "hip/hip_runtime.h"
#include <neuralnetwork/SigmoidLayer.h>

SigmoidLayer::SigmoidLayer(std::string name)
{
  this->name = name;
}

Matrix SigmoidLayer::forward(const Matrix& X)
{
  Matrix res = X;
  Matrix e = X;

  for(int i = 0; i < res.getRows(); i++)
  {
    for(int j = 0; j < res.getCols(); j++)
    {
      if (res(i,j) > 10)
      {
        res(i,j) = 10;
      }
      else if (res(i, j) < -10)
      {
        res(i, j) = -10;
      }
      e(i, j) = exp(res(i,j));
      res(i, j) = e(i,j)+1;
      res(i, j) = e(i, j)/res(i,j);

    }
  }
  return res;
}

Matrix SigmoidLayer::backprop(Matrix& dZ, const Matrix& X, float learning_rate)
{
  // o(x)*(1-o(x))
  Matrix m1(X.getRows(), X.getCols(), 1.0f);
  Matrix ox = this->forward(dZ);
  m1 = m1 - ox;
  m1 = ox.hadamard(m1);
  return m1;
}